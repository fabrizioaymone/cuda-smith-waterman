#include "hip/hip_runtime.h"
#include "gpu_conventional.cuh"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>

#define S_LEN 512
#define DIM S_LEN+1
#define N 1000

#define CHECK(call)                                                                       \
    {                                                                                     \
        const hipError_t err = call;                                                     \
        if (err != hipSuccess)                                                           \
        {                                                                                 \
            printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
            exit(EXIT_FAILURE);                                                           \
        }                                                                                 \
    }

#define CHECK_KERNELCALL()                                                                \
    {                                                                                     \
        const hipError_t err = hipGetLastError();                                       \
        if (err != hipSuccess)                                                           \
        {                                                                                 \
            printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__); \
            exit(EXIT_FAILURE);                                                           \
        }                                                                                 \
    }


double get_time_gpu() // function to get the time of day in seconds
{
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return tv.tv_sec + tv.tv_usec * 1e-6;
}


__device__ int max4_gpu(int n1, int n2, int n3, int n4) {
    int tmp1, tmp2;
    tmp1 = n1 > n2 ? n1 : n2;
    tmp2 = n3 > n4 ? n3 : n4;
    tmp1 = tmp1 > tmp2 ? tmp1 : tmp2;
    return tmp1;
}


__device__ void backtrace_gpu(char *simple_rev_cigar, char dir_mat[], int i, int j, int max_cigar_len) {
    int k;
    for (k = 0; k < max_cigar_len && dir_mat[i * (S_LEN + 1) + j] != 0; k++) {
        int dir = dir_mat[i * (S_LEN + 1) + j];
        if (dir == 1 || dir == 2) {
            i--;
            j--;
        } else if (dir == 3)
            i--;
        else if (dir == 4)
            j--;

        simple_rev_cigar[k] = dir;
        // if(k==0 && )
    }
}


__global__ void diagonal_kernel(char *query, char *reference, int *res, char *simple_rev_cigar, char *dir_mat_gpu) {
    int threadId = threadIdx.x;
    int n = blockIdx.x;
    char *dir_mat = &dir_mat_gpu[n * (S_LEN + 1) * (S_LEN + 1)];

    __shared__ char shared_query[S_LEN];
    __shared__ char shared_reference[S_LEN];

    if (threadId < S_LEN)
        shared_query[threadId] = query[n * S_LEN + threadId];
    else if (threadId < S_LEN * 2)
        shared_reference[threadId - S_LEN] = reference[n * S_LEN + (threadId - S_LEN)];

   // __syncthreads();

    __shared__ int mat[S_LEN + 1];
    __shared__ int prev[S_LEN + 1];
    __shared__ int prev2[S_LEN + 1];

    //matrix initialisation
    prev[0] = 0;
    prev[1] = 0;
    prev2[0] = 0;
   // __syncthreads();

    int ins = -2, del = -2, match = 1, mismatch = -1; // penalties
    __shared__ int maxi, maxj;
    __shared__ int max;
    max = -1;
    __shared__ int threadMax;

    for (int diag = 2; diag < (2 * S_LEN + 1); diag++) {
        int length = DIM - abs(DIM - (diag + 1));
        int length_1 = DIM - abs(DIM - (diag));
        int length_2 = DIM - abs(DIM - (diag - 1));
        if (DIM > diag) {
            mat[0] = 0;
            mat[length - 1] = 0;
        }
        threadMax = -1;

        __syncthreads();

        int row = (DIM > diag) ? (diag - threadId) : ((DIM - 1) - threadId);
        int col = (DIM > diag) ? threadId : ((diag - (DIM - 1)) + threadId);

        int tmp = -2;
        if (threadId < length) {
            if (row != 0 && col != 0) {
                // compare the sequences characters

                int comparison = (shared_query[row - 1] == shared_reference[col - 1]) ? match : mismatch;

                // compute the cell knowing the comparison result


                tmp = max4_gpu(prev2[threadId + (length_2 - length) / 2] + comparison,
                               prev[(DIM > diag) ? threadId : (threadId + 1)] + del,
                               prev[(DIM > diag) ? (threadId - 1) : threadId] + ins,
                               0);
                //max4(sc_mat[i - 1][j - 1] + comparison, sc_mat[i - 1][j] + del, sc_mat[i][j - 1] + ins, 0);

                char dir;

                if (tmp == (prev2[threadId + (length_2 - length) / 2] + comparison))
                    dir = comparison == match ? 1 : 2;
                else if (tmp == (prev[(DIM > diag) ? threadId : (threadId + 1)] + del))
                    dir = 3;
                else if (tmp == (prev[(DIM > diag) ? (threadId - 1) : threadId] + ins))
                    dir = 4;
                else
                    dir = 0;

                dir_mat[row * (S_LEN + 1) + col] = dir;
                mat[threadId] = tmp;
                atomicMax(&max, tmp);

            }
        }
        __syncthreads(); // se non sincronizzi hai thread che non entrando nell'if sopra ti modificano prev e prev2 prima che i thread nell'if li usino per calcolare la score
        if (threadId < length_1)
            prev2[threadId] = prev[threadId];
        if (threadId < length)
            prev[threadId] = mat[threadId];

        __syncthreads();

         if (tmp == max) {
            atomicMax(&threadMax, threadId);
         }
        __syncthreads();

        if (tmp == max && threadId == threadMax) {
            maxi = row;
            maxj = col;
        }
        __syncthreads();
    }

    __syncthreads();
    res[n] = max;

    if (threadId == 0)
        backtrace_gpu(&simple_rev_cigar[n * S_LEN * 2], dir_mat, maxi, maxj, S_LEN * 2);

}


int gpu_conventional(char **query, char **reference, int **sc_mat, char **dir_mat, int *res, char **simple_rev_cigar) {

    //declare global variables


    char *query_gpu;
    char *reference_gpu;
    int *res_gpu;
    char *simple_rev_cigar_gpu;
    char *dir_mat_gpu;

    int *res_host = (int *) malloc(N * sizeof(int));

    char *simple_rev_cigar_host = (char *) malloc(N * S_LEN * 2 * sizeof(char));

    //allocate space in global memory and copy original ones -> linearize matrices!!

    CHECK(hipMalloc((void **) &query_gpu, N * S_LEN * sizeof(char)));
    for (int i = 0; i < N; i++) CHECK(
            hipMemcpy(&query_gpu[i * S_LEN], query[i], S_LEN * sizeof(char), hipMemcpyHostToDevice));

    CHECK(hipMalloc((void **) &reference_gpu, N * S_LEN * sizeof(char)));
    for (int i = 0; i < N; i++) CHECK(
            hipMemcpy(&reference_gpu[i * S_LEN], reference[i], S_LEN * sizeof(char), hipMemcpyHostToDevice));

    CHECK(hipMalloc((void **) &res_gpu, N * sizeof(int)));

    CHECK(hipMalloc((void **) &simple_rev_cigar_gpu, N * S_LEN * 2 * sizeof(char)));

    CHECK(hipMalloc((void **) &dir_mat_gpu, N * (S_LEN + 1) * (S_LEN + 1) * sizeof(char)));
    CHECK(hipMemset((void *) dir_mat_gpu, 0, N * (S_LEN + 1) * (S_LEN + 1) * sizeof(char)));

    dim3 gridsize(1000);
    dim3 blocksize(1024);


    double start_gpu = get_time_gpu();

    diagonal_kernel<<<gridsize, blocksize>>>(query_gpu, reference_gpu, res_gpu, simple_rev_cigar_gpu, dir_mat_gpu);
    CHECK_KERNELCALL();


    CHECK(hipDeviceSynchronize());
    double end_gpu = get_time_gpu();

    printf("SW Time GPU: %.10lf\n", end_gpu - start_gpu);

    CHECK(hipMemcpy(res_host, res_gpu, N * sizeof(int), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(simple_rev_cigar_host, simple_rev_cigar_gpu, N * S_LEN * 2 * sizeof(char),
                     hipMemcpyDeviceToHost));





    // verify res are correct
    for (int i = 0; i < N; i++)
        if (res_host[i] != res[i]) {
            printf("ERROR! N: %d res: %d res_host: %d\n", i, res[i], res_host[i]);
        }

    // verify cigars are correct
    for (int i = 0; i < N; i++) {
        int flag = 0;
        for (int j = 0; j < S_LEN * 2 && !flag; j++) {

            if (simple_rev_cigar[i][j] != simple_rev_cigar_host[i * S_LEN * 2 + j]) {
                printf("ERROR! N: %d index:%d cpu_cigar: %d gpu_cigar: %d\n", i, j, simple_rev_cigar[i][j],
                       simple_rev_cigar_host[i * S_LEN * 2 + j]);
                flag = 1;
            }
        }
    }

    return 0;
}

